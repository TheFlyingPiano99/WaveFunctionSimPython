#include "hip/hip_runtime.h"
#include "PATH_TO_SOURCES/cuda_kernels/common.cu"


extern "C" __global__
void draining_potential_kernel(
    complex<float>* __restrict__ V,

    float delta_x,
    float delta_y,
    float delta_z,

    float ellipsoid_a,
    float ellipsoid_b,
    float ellipsoid_c,

    float inner_ellipsoid_distance_bohr_radii,

    float max_potential_hartree,
    float exponent
)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    int idx = i * gridDim.x * blockDim.x * gridDim.y * blockDim.y
            + j * gridDim.x * blockDim.x
            + k;

    float3 delta_r = {delta_x, delta_y, delta_z};
    float3 N = {(float)(gridDim.x * blockDim.x), (float)(gridDim.y * blockDim.y), (float)(gridDim.z * blockDim.z)};
    float3 pos = diff(
        mul(delta_r, {k, j, i}),
        scalarVectorMul(0.5f, mul(N, delta_r))
    );
    float ellipsoid_distance =
        pos.x * pos.x / ellipsoid_a / ellipsoid_a
        + pos.y * pos.y / ellipsoid_b / ellipsoid_b
        + pos.z * pos.z / ellipsoid_c / ellipsoid_c
        - 1.0f;     // The implicit equation of the ellipsoid

    float t = fmaxf(
            0.0f,
            ellipsoid_distance - inner_ellipsoid_distance_bohr_radii
        ) / -inner_ellipsoid_distance_bohr_radii;
    V[idx] += complex<float>(0.0f, powf(t, exponent) * max_potential_hartree);
}
