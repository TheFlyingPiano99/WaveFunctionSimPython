#include "hip/hip_runtime.h"
#include "PATH_TO_SOURCES/cuda_kernels/common.cu"


extern "C" __global__
void kinetic_operator_kernel(
    complex<float>* __restrict__ kinetic_operator,

    float delta_x,
    float delta_y,
    float delta_z,

    float delta_t
)
{
    uint3 voxel = get_voxel_coords();
    int idx = get_array_index();

    float3 f = div(
        {(float)voxel.x, (float)voxel.y, (float)voxel.z},
        {(float)(gridDim.x * blockDim.x - 1), (float)(gridDim.y * blockDim.y - 1), (float)(gridDim.z * blockDim.z - 1)}
    );
    float3 delta_r = {delta_x, delta_y, delta_z};

    // Account for numpy fftn's "negative frequency in second half" pattern
    if (f.x > 0.5f)
        f.x = 1.0f - f.x;
    if (f.y > 0.5f)
        f.y = 1.0f - f.y;
    if (f.z > 0.5f)
        f.z = 1.0f - f.z;

    float3 momentum = scalarVectorMul(2.0f * M_PI, div(f, delta_r));
    float angle = -dot(momentum, momentum) * delta_t / 4.0f;
    kinetic_operator[idx] = exp_i(angle);
}
