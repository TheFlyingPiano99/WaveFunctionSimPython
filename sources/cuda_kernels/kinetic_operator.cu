#include "hip/hip_runtime.h"
#include "PATH_TO_SOURCES/cuda_kernels/common.cu"


extern "C" __global__
void kinetic_operator_kernel(
    complex<float>* __restrict__ kinetic_operator,

    float delta_x,
    float delta_y,
    float delta_z,

    float delta_t
)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;
    int idx = i * gridDim.x * blockDim.x * gridDim.y * blockDim.y
            + j * gridDim.x * blockDim.x
            + k;

    float3 f = div(
        {(float)k, (float)j, (float)i},
        {(float)(gridDim.x * blockDim.x - 1), (float)(gridDim.y * blockDim.y - 1), (float)(gridDim.z * blockDim.z - 1)}
    );
    float3 delta_r = {delta_x, delta_y, delta_z};

    // Account for numpy fftn's "negative frequency in second half" pattern
    if (f.x > 0.5f)
        f.x = 1.0f - f.x;
    if (f.y > 0.5f)
        f.y = 1.0f - f.y;
    if (f.z > 0.5f)
        f.z = 1.0f - f.z;

    float3 momentum = scalarVectorMul(2.0f * M_PI, div(f, delta_r));
    float angle = -dot(momentum, momentum) * delta_t / 4.0f;
    kinetic_operator[idx] = exp_i(angle);
}
