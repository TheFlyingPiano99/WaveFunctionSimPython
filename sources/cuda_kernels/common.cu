#include "hip/hip_runtime.h"
#ifndef CUDA_COMMON_INCLUDE
#define CUDA_COMMON_INCLUDE

#include <cupy/complex.cuh>


constexpr float M_PI = 3.14159265359;

__device__ constexpr float3 scalarVectorMul(const float s, const float3& v)
{
    return {s * v.x, s * v.y, s * v.z};
}

constexpr __device__ float dot(const float3& a, const float3& b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

constexpr __device__ float3 cross(const float3& a, const float3& b)
{
    return {
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    };
}

__device__ const complex<float> exp_i(float angle)
{
    return complex<float>(cosf(angle), sinf(angle));
}

__device__ const complex<float> cexp_i(const complex<float>& cangle)
{
    return complex<float>(cosf(cangle.real()), sinf(cangle.real())) * expf(-cangle.imag());
}

__device__ constexpr float3 add(const float3& a, const float3& b)
{
    return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ constexpr float3 diff(const float3& a, const float3& b)
{
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ float3 mul(const float3& a, const float3 b)
{
    return {a.x * b.x, a.y * b.y, a.z * b.z};
}

__device__ constexpr float3 div(const float3& a, const float3& b)
{
    return {a.x / b.x, a.y / b.y, a.z / b.z};
}

__device__ float3 normalize(const float3& a)
{
    return scalarVectorMul(1.0f / sqrtf(dot(a, a)), a);
}

__device__ constexpr float3 transform_corner_origin_to_center_origin_system(const float3& pos)
{
    return diff(
        pos,
        scalarVectorMul(0.5f, {(float)(gridDim.x * blockDim.x), (float)(gridDim.y * blockDim.y), (float)(gridDim.z * blockDim.z)})
    );
}

__device__ uint3 get_voxel_coords()
{
    return {
        blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y,
        blockIdx.z * blockDim.z + threadIdx.z
    };
}

__device__ uint3 get_voxel_coords_inverted()
{
    unsigned int x = gridDim.x * blockDim.x - (blockIdx.x * blockDim.x + threadIdx.x) - 1;
    unsigned int y = gridDim.y * blockDim.y - (blockIdx.y * blockDim.y + threadIdx.y) - 1;
    unsigned int z = gridDim.z * blockDim.z - (blockIdx.z * blockDim.z + threadIdx.z) - 1;
    return {x, y, z};
}

__device__ int get_array_index()
{
    uint3 voxel = get_voxel_coords();
    return voxel.x * gridDim.y * blockDim.y * gridDim.z * blockDim.z
            + voxel.y * gridDim.z * blockDim.z
            + voxel.z;
}

__device__ int get_array_index_inverted()
{
    uint3 voxel = get_voxel_coords_inverted();
    return voxel.x * gridDim.y * blockDim.y * gridDim.z * blockDim.z
            + voxel.y * gridDim.z * blockDim.z
            + voxel.z;
}


__device__ float3 mat3x3_vector_mul(const float (&m)[3][3], const float3& v)
{
    return {
        m[0][0] * v.x + m[0][1] * v.y + m[0][2] * v.z,
        m[1][0] * v.x + m[1][1] * v.y + m[1][2] * v.z,
        m[2][0] * v.x + m[2][1] * v.y + m[2][2] * v.z
    };
}


__device__ float3 rotate_vector(const float3& v, const float3& axis, float rad)
{
    float q0 = cosf(rad / 2.0f);
    float q1 = sinf(rad / 2.0f) * axis.x;
    float q2 = sinf(rad / 2.0f) * axis.y;
    float q3 = sinf(rad / 2.0f) * axis.z;
    float Q[3][3] = { { 0.0f } }; // 3x3 rotation matrix

    Q[0][0] = q0*q0 + q1*q1 - q2*q2 - q3*q3;
    Q[0][1] = 2.0f * (q1*q2 - q0*q3);
    Q[0][2] = 2.0f * (q1*q3 + q0*q2);

    Q[1][0] = 2.0f * (q2*q1 + q0*q3);
    Q[1][1] = q0*q0 - q1*q1 + q2*q2 - q3*q3;
    Q[1][2] = 2.0f * (q2*q3 - q0*q1);

    Q[2][0] = 2.0f * (q3*q1 - q0*q2);
    Q[2][1] = 2.0f * (q3*q2 + q0*q1);
    Q[2][2] = q0*q0 - q1*q1 - q2*q2 + q3*q3;

    return mat3x3_vector_mul(Q, v);
    //return v;
}

#endif  // CUDA_COMMON
