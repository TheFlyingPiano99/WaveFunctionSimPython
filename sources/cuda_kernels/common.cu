#include "hip/hip_runtime.h"
#ifndef CUDA_COMMON_INCLUDE
#define CUDA_COMMON_INCLUDE

#include <cupy/complex.cuh>


constexpr float M_PI = 3.14159265359;

__device__ float3 fabsf(const float3& v)
{
    return {fabsf(v.x), fabsf(v.y), fabsf(v.z)};
}

__device__ complex<float> conj(const complex<float>& c)
{
    return complex<float>(c.real(), -c.imag());
}

__device__ complex<float> mul(const complex<float>& a, const complex<float>& b)
{
    return complex<float>(a.real() * b.real() - a.imag() * b.imag(), a.real() * b.imag() + a.imag() * b.real() );
}

__device__ constexpr float3 scalarVectorMul(const float s, const float3& v)
{
    return {s * v.x, s * v.y, s * v.z};
}

__device__ constexpr float3 operator*(const float s, const float3& v)
{
    return {s * v.x, s * v.y, s * v.z};
}

__device__ constexpr float3 operator*(const float3& v, const float s)
{
    return {s * v.x, s * v.y, s * v.z};
}

__device__ constexpr float3 operator-(const float3& v)
{
    return {-v.x, -v.y, -v.z};
}

constexpr __device__ float dot(const float3& a, const float3& b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

constexpr __device__ float3 cross(const float3& a, const float3& b)
{
    return {
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    };
}

__device__ const complex<float> exp_i(float angle)
{
    return complex<float>(cosf(angle), sinf(angle));
}

__device__ const complex<float> cexp_i(const complex<float>& cangle)
{
    return complex<float>(cosf(cangle.real()), sinf(cangle.real())) * expf(-cangle.imag());
}

__device__ constexpr float3 add(const float3& a, const float3& b)
{
    return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ constexpr float3 operator+(const float3& a, const float3& b)
{
    return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ constexpr float3 diff(const float3& a, const float3& b)
{
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ constexpr float3 operator-(const float3& a, const float3& b)
{
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ float3 mul(const float3& a, const float3 b)
{
    return {a.x * b.x, a.y * b.y, a.z * b.z};
}

__device__ float3 operator*(const float3& a, const float3 b)
{
    return {a.x * b.x, a.y * b.y, a.z * b.z};
}

__device__ constexpr float3 div(const float3& a, const float3& b)
{
    return {a.x / b.x, a.y / b.y, a.z / b.z};
}

__device__ constexpr float3 operator/(const float3& a, const float3& b)
{
    return {a.x / b.x, a.y / b.y, a.z / b.z};
}

__device__ float length(const float3& a)
{
    return sqrtf(dot(a, a));
}

__device__ float3 normalize(const float3& a)
{
    return scalarVectorMul(1.0f / sqrtf(dot(a, a)), a);
}

__device__ constexpr float3 transform_corner_origin_to_center_origin_system(const float3& pos)
{
    return diff(
        pos,
        scalarVectorMul(0.5f, {(float)(gridDim.x * blockDim.x), (float)(gridDim.y * blockDim.y), (float)(gridDim.z * blockDim.z)})
    );
}

__device__ uint3 get_voxel_count()
{
    return {
        gridDim.x * blockDim.x,
        gridDim.y * blockDim.y,
        gridDim.z * blockDim.z
    };
}

__device__ uint3 get_voxel_coords()
{
    return {
        blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y,
        blockIdx.z * blockDim.z + threadIdx.z
    };
}

__device__ uint3 get_voxel_coords_inverted()
{
    unsigned int x = gridDim.x * blockDim.x - (blockIdx.x * blockDim.x + threadIdx.x) - 1;
    unsigned int y = gridDim.y * blockDim.y - (blockIdx.y * blockDim.y + threadIdx.y) - 1;
    unsigned int z = gridDim.z * blockDim.z - (blockIdx.z * blockDim.z + threadIdx.z) - 1;
    return {x, y, z};
}

__device__ int get_array_index()
{
    uint3 voxel = get_voxel_coords();
    return voxel.x * gridDim.y * blockDim.y * gridDim.z * blockDim.z
            + voxel.y * gridDim.z * blockDim.z
            + voxel.z;
}

__device__ int get_array_index(const uint3& voxel)
{
    return voxel.x * gridDim.y * blockDim.y * gridDim.z * blockDim.z
            + voxel.y * gridDim.z * blockDim.z
            + voxel.z;
}

__device__ int get_array_index(const uint3& voxel, const uint3& N)
{
    return voxel.x * N.y * N.z
            + voxel.y * N.z
            + voxel.z;
}

__device__ int get_array_index_inverted()
{
    uint3 voxel = get_voxel_coords_inverted();
    return voxel.x * gridDim.y * blockDim.y * gridDim.z * blockDim.z
            + voxel.y * gridDim.z * blockDim.z
            + voxel.z;
}


__device__ float3 operator*(const float (&m)[3][3], const float3& v)
{
    return {
        m[0][0] * v.x + m[0][1] * v.y + m[0][2] * v.z,
        m[1][0] * v.x + m[1][1] * v.y + m[1][2] * v.z,
        m[2][0] * v.x + m[2][1] * v.y + m[2][2] * v.z
    };
}

__device__ float4 operator*(const float (&m)[4][4], const float4& v)
{
    return {
        m[0][0] * v.x + m[0][1] * v.y + m[0][2] * v.z + m[0][3] * v.w,
        m[1][0] * v.x + m[1][1] * v.y + m[1][2] * v.z + m[1][3] * v.w,
        m[2][0] * v.x + m[2][1] * v.y + m[2][2] * v.z + m[2][3] * v.w,
        m[3][0] * v.x + m[3][1] * v.y + m[3][2] * v.z + m[3][3] * v.w,
    };
}

__device__ float3 rotate_vector(const float3& v, const float3& axis, float rad)
{
    float q0 = cosf(rad / 2.0f);
    float q1 = sinf(rad / 2.0f) * axis.x;
    float q2 = sinf(rad / 2.0f) * axis.y;
    float q3 = sinf(rad / 2.0f) * axis.z;
    float Q[3][3] = { { 0.0f } }; // 3x3 rotation matrix

    Q[0][0] = q0*q0 + q1*q1 - q2*q2 - q3*q3;
    Q[0][1] = 2.0f * (q1*q2 - q0*q3);
    Q[0][2] = 2.0f * (q1*q3 + q0*q2);

    Q[1][0] = 2.0f * (q2*q1 + q0*q3);
    Q[1][1] = q0*q0 - q1*q1 + q2*q2 - q3*q3;
    Q[1][2] = 2.0f * (q2*q3 - q0*q1);

    Q[2][0] = 2.0f * (q3*q1 - q0*q2);
    Q[2][1] = 2.0f * (q3*q2 + q0*q1);
    Q[2][2] = q0*q0 - q1*q1 - q2*q2 + q3*q3;

    return Q * v;
}

__device__ float mix(const float3& xyz, float u, float v)
{
    return xyz.z * v + (1.0f - v) * (xyz.y * u + xyz.x * (1.0f - u));
}

#endif  // CUDA_COMMON
