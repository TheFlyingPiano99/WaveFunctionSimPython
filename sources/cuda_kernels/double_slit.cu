#include "hip/hip_runtime.h"
#include "PATH_TO_SOURCES/cuda_kernels/common.cu"

extern "C" __global__
void double_slit_kernel(
    complex<float>* __restrict__ V,

    float delta_x,
    float delta_y,
    float delta_z,

    float center_x,
    float center_y,
    float center_z,

    float thickness_bohr_radii,
    float potential_hartree,
    float space_between_slits_bohr_radii,
    float slit_width_bohr_radii
)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    int idx = i * gridDim.x * blockDim.x * gridDim.y * blockDim.y
            + j * gridDim.x * blockDim.x
            + k;

    float3 center = {center_x, center_y, center_z};
    float3 delta_r = {delta_x, delta_y, delta_z};
    float3 r = delta_r * transform_corner_origin_to_center_origin_system({(float)i, (float)j, (float)k});

    if (
        r.x > center.x - thickness_bohr_radii / 2.0f
        && r.x < center.x + thickness_bohr_radii / 2.0f
        && !(
            (
                r.z
                > center.z
                - space_between_slits_bohr_radii * 0.5f
                - slit_width_bohr_radii
                && r.z
                < center.z
                - space_between_slits_bohr_radii * 0.5f
            )
            or (
                r.z
                < center.z
                + space_between_slits_bohr_radii * 0.5f
                + slit_width_bohr_radii
                and r.z
                > center.z
                + space_between_slits_bohr_radii * 0.5f
            )
        )
    ) {
        V[idx] += potential_hartree;
    }

}
