#include "hip/hip_runtime.h"
#include "PATH_TO_SOURCES/cuda_kernels/common.cu"

extern "C" __global__
void double_slit_kernel(
    complex<float>* __restrict__ V,

    float delta_x,
    float delta_y,
    float delta_z,

    float center_x,
    float center_y,
    float center_z,

    float thickness_bohr_radii,
    float potential_hartree,
    float space_between_slits_bohr_radii,
    float slit_width_bohr_radii
)
{
    uint3 voxel = get_voxel_coords();
    int idx = get_array_index();

    float3 center = {center_x, center_y, center_z};
    float3 delta_r = {delta_x, delta_y, delta_z};
    float3 r = delta_r * transform_corner_origin_to_center_origin_system({(float)voxel.x, (float)voxel.y, (float)voxel.z});

    if (
        r.x > center.x - thickness_bohr_radii / 2.0f
        && r.x < center.x + thickness_bohr_radii / 2.0f
        && !(
            (
                r.z
                > center.z
                - space_between_slits_bohr_radii * 0.5f
                - slit_width_bohr_radii
                && r.z
                < center.z
                - space_between_slits_bohr_radii * 0.5f
            )
            or (
                r.z
                < center.z
                + space_between_slits_bohr_radii * 0.5f
                + slit_width_bohr_radii
                and r.z
                > center.z
                + space_between_slits_bohr_radii * 0.5f
            )
        )
    ) {
        V[idx] += potential_hartree;
    }

}
