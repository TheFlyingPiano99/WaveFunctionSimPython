#include "hip/hip_runtime.h"
#include "PATH_TO_SOURCES/cuda_kernels/common.cu"


extern "C" __global__
void potential_wall_kernel(
    complex<float>* __restrict__ V,

    float delta_x,
    float delta_y,
    float delta_z,

    float center_x,
    float center_y,
    float center_z,

    float normal_x,
    float normal_y,
    float normal_z,

    float thickness_bohr_radius,
    float potential_hartree
)
{
    uint3 voxel = get_voxel_coords();
    int idx = get_array_index();

    float3 center = {center_x, center_y, center_z};
    float3 normal = {normal_x, normal_y, normal_z};
    float3 delta_r = {delta_x, delta_y, delta_z};

    float3 r = mul(delta_r, transform_corner_origin_to_center_origin_system({(float)voxel.x, (float)voxel.y, (float)voxel.z}));
    float d = dot(normal, diff(center, r));
    if (d <= thickness_bohr_radius * 0.5f && d >= -thickness_bohr_radius * 0.5f)
    {
        V[idx] += potential_hartree * (1.0f - fmaxf(2.0f * fabsf(d / thickness_bohr_radius * 2.0f) - 1.0, 0.0f));
    }
}
